#include <iostream>
#include <hip/hip_runtime.h>

// Kernel function to add two arrays element-wise
__global__ void addArrays(float* a, float* b, float* result, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        result[index] = a[index] + b[index];
    }
}

// Function to add two arrays on the GPU
void addArraysOnGPU(float* hostArrayA, float* hostArrayB, float* hostResult, int arraySize) {

    // std::cout << "Array A" << std::endl;

    // for (int i = 0; i < arraySize; ++i) {
    //     std::cout << hostArrayA[i] << " ";
    // }

    // Device arrays
    float* deviceArrayA;
    float* deviceArrayB;
    float* deviceResult;

    // Allocate memory on the GPU
    hipMalloc((void**)&deviceArrayA, arraySize * sizeof(float));
    hipMalloc((void**)&deviceArrayB, arraySize * sizeof(float));
    hipMalloc((void**)&deviceResult, arraySize * sizeof(float));

    // Copy data from host to device
    hipMemcpy(deviceArrayA, hostArrayA, arraySize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceArrayB, hostArrayB, arraySize * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int blockSize = 256;
    int gridSize = (arraySize + blockSize - 1) / blockSize;

    // Launch the kernel
    addArrays<<<gridSize, blockSize>>>(deviceArrayA, deviceArrayB, deviceResult, arraySize);

    // Copy the result back to the host
    hipMemcpy(hostResult, deviceResult, arraySize * sizeof(float), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(deviceArrayA);
    hipFree(deviceArrayB);
    hipFree(deviceResult);
}
