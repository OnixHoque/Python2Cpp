
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel(){
  printf("hello\n");
}

void gpu_fw(){
  mykernel<<<1,1>>>();
  hipDeviceSynchronize();
}
