#include "kernels"

void cuda_spmm_test()
{
    // Allocate host (CPU) memory
    int64_t *ptrb = new int64_t[5]{0, 1, 2, 3, 4};
    int64_t *indx = new int64_t[4]{0, 1, 2, 3}; 
    float *val = new float[4]{1.0, 2.0, 3.0, 4.0};

    float mat[4][4] = {
	    		{2.0, 2.0, 2.0, 2.0},
	    		{2.0, 2.0, 2.0, 2.0},
	    		{2.0, 2.0, 2.0, 2.0},
	    		{2.0, 2.0, 2.0, 2.0}};
    float out[4][4] = {
                        {0, 0, 0, 0},
                        {0, 0, 0, 0},
                        {0, 0, 0, 0},
                        {0, 0, 0, 0}};
    int m=4;
    int n=4;
    int k=4;
    int nnz=4;
    int rows=m;
    int cols=n;

    // Allocate device (GPU) memory
    int64_t *ptrb_device, *indx_device; 
    float *val_device, *mat_device, *out_device;
    hipMalloc(&ptrb_device, (m+1) * sizeof(int64_t));
    hipMalloc(&indx_device, nnz * sizeof(int64_t));
    hipMalloc(&val_device, nnz * sizeof(float));
    hipMalloc(&mat_device, n * k * sizeof(float));
    hipMalloc(&out_device, m * k * sizeof(float)); 
    
    // Copy input data from host to device
    hipMemcpy(ptrb_device, ptrb, (m+1) * sizeof(int64_t), hipMemcpyHostToDevice);
    hipMemcpy(indx_device, indx, nnz * sizeof(int64_t), hipMemcpyHostToDevice);
    hipMemcpy(val_device, val, nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(mat_device, mat, n * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out_device, out, m * k * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    fusedmm_cuda(m, n, k, nnz, indx_device, ptrb_device, val_device, mat_device, out_device);
	
    // Copy output data from device to host
    hipMemcpy(out, out_device, m * k * sizeof(float), hipMemcpyDeviceToHost);
    
    // Print output
    for (int i = 0; i < m; i++) {
	    for(int j=0; j<k; j++){
        	std::cout << out[i][j] << " ";
	    }
	    std::cout << "\n";
    }
    std::cout << std::endl;

    // Free memory
    hipFree(ptrb_device);
    hipFree(indx_device);
    hipFree(val_device);
    hipFree(mat_device);
    hipFree(out_device);

    //return 0;
}
